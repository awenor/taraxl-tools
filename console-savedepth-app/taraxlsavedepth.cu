#include <iostream>
#include <ctime>
#include  <iomanip>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>

#include "TaraXL.h"
#include "TaraXLCam.h"
#include "TaraXLDepth.h"

using namespace std;
using namespace cv;
using namespace TaraXLSDK;

int main () {

  TaraXL taraxlCam;
  TaraXLCam selectedCam;
  TaraXLCamList taraxlCamList;
  ResolutionList supportedResolutions;
  TaraXLDepth *taraxlDepth;

  uint camIndex, iResIndex, iAccuracyMode;
  TARAXL_STATUS_CODE status;

  status = taraxlCam.enumerateDevices(taraxlCamList);
  if (status != TARAXL_SUCCESS) {

      cout << "Camera enumeration failed" << endl;
      return 1;
  }

  if (taraxlCamList.size() == 0) {

    cout << "No cameras connected" << endl;
    return 1;
  }

  cout << "Select a device:" << endl;
  for (int i = 0; i < taraxlCamList.size(); i++) {

    cout << i << ": ";
    string name;
    taraxlCamList[i].getFriendlyName(name);
    cout << name << endl;
  }

  cin >> camIndex;
  if (cin.fail()) {

    cout << "Invalid input" << endl;
    return 1;
  }

  if (camIndex >= taraxlCamList.size()) {

    cout << "Invalid input" << endl;
    return 1;
  }

  selectedCam = taraxlCamList.at(camIndex);

  status = selectedCam.connect();
  if (status != TARAXL_SUCCESS) {

      cout << "Camera connect failed" << endl;
      return 1;
  }

  status = selectedCam.getResolutionList(supportedResolutions);
  if (status != TARAXL_SUCCESS) {

      cout << "Get camera resolutions failed" << endl;
      return 1;
  }

  cout << endl << "Select a resolution:" << endl;
  for (int i = 0; i < supportedResolutions.size(); i++) {

    cout << i << ": ";
    string resolution = "";
    resolution += to_string(supportedResolutions[i].width);
    resolution += " x ";
    resolution += to_string(supportedResolutions[i].height);
    cout << resolution << endl;
  }
  cin >> iResIndex;

  if (cin.fail()) {

    cout << "Invalid input" << endl;
    return 1;
  }

  if (iResIndex >= supportedResolutions.size()) {

    cout << "Invalid input" << endl;
    return 1;
  }

  status = selectedCam.setResolution(supportedResolutions.at(iResIndex));
  if (status != TARAXL_SUCCESS) {

      cout << "Set resolutions failed" << endl;
      return 1;
  }

  taraxlDepth = new TaraXLDepth(selectedCam);
  if (taraxlDepth == NULL) {

    cout << "Unable to create instance to TaraDepth" << endl;
    return 1;
  }

  cout << endl << "Select a Accuracy mode:" << endl;
  cout << "0: High Accuracy" << endl << "1: Low Accuracy" << endl;
  cin >> iAccuracyMode;

  if (cin.fail()) {

    cout << "Invalid input" << endl;
    return 1;
  }

  if (iAccuracyMode == 0) {

    status = taraxlDepth->setAccuracy(HIGH);
  }
  else if (iAccuracyMode == 1) {

    status = taraxlDepth->setAccuracy(LOW);
  }
  else {

    cout << "Invalid input" << endl;
    return 1;
  }

  if (status != TARAXL_SUCCESS) {

      cout << "Set accuracy failed" << endl;
  }

  Mat left, right, grayDisp, colorDisp, depthMap;
  status = taraxlDepth->getMap(left, right, grayDisp, true, depthMap, true);
  if (status != TARAXL_SUCCESS) {

      cout << "Get map failed" << endl;
      delete taraxlDepth;
      return 1;
  }

  applyColorMap(grayDisp, colorDisp, COLORMAP_JET);

  imwrite("../left.jpg", left);
  imwrite("../right.jpg", right);
  imwrite("../grayDisparity.jpg", grayDisp);
  imwrite("../colorDisparity.jpg", colorDisp);
  imwrite("../depthMap.jpg", depthMap);

  cout << endl << "Images saved to the disk!!!!" << endl;

  delete taraxlDepth;
  exit(0);
}
