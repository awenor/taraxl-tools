#include <iostream>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>

#include "TaraXL.h"
#include "TaraXLCam.h"
#include "TaraXLPointcloud.h"

#include "boost/thread/thread.hpp"
#include <boost/thread/mutex.hpp>
#include "pcl/common/common_headers.h"
#include "pcl/visualization/pcl_visualizer.h"
#include "pcl/visualization/cloud_viewer.h"
#include <pcl/visualization/common/common.h>
#include "pcl/common/transforms.h"

using namespace std;
using namespace cv;
using namespace TaraXLSDK;
using namespace pcl;


enum quality{STANDARD = 0, MEDIUM = 1, HIGHEST = 2 };
quality pcl_quality = quality::HIGHEST;
enum save{PLY = 0, PCD = 1, VTK = 2};
save pcl_save = PLY;
int showSaved=0,savePressed=0;

boost::mutex ioMutex;


void viewerOneOff (pcl::visualization::PCLVisualizer& viewer)
{

	viewer.setShowFPS(false);
    viewer.addText("Quality mode : Highest",20,80,15,10,10,255,"mode");
    viewer.addText("Press m/M to change quality",20,60,15,10,10,255,"modeInst");
    viewer.addText("Save format : PLY",20,40,15,10,10,255,"saveFormat");
    viewer.addText("Press n/N to change save format",20,20,15,10,10,255,"saveInst");
    viewer.addText("Press SHIFT+S to save",20,0,15,10,10,255,"saved");

}


void viewerUpdate (pcl::visualization::PCLVisualizer& viewer)
{

	if(pcl_quality==quality::STANDARD)
		viewer.updateText("Quality mode : Standard",20,80,15,10,10,255,"mode");
    if(pcl_quality==quality::MEDIUM)
        viewer.updateText("Quality mode : Medium",20,80,15,10,10,255,"mode");
    if(pcl_quality==quality::HIGHEST)
        viewer.updateText("Quality mode : Highest",20,80,15,10,10,255,"mode");


    if(pcl_save==PLY)
		viewer.updateText("Save format : PLY",20,40,15,10,10,255,"saveFormat");
    if(pcl_save==PCD)
        viewer.updateText("Save format : PCD",20,40,15,10,10,255,"saveFormat");
    if(pcl_save==VTK)
        viewer.updateText("Save format : VTK",20,40,15,10,10,255,"saveFormat");

    if(savePressed==1)
    {
		viewer.updateText("Saved",20,0,15,10,10,255,"saved");
	    showSaved++;
     	if(showSaved>10)
     	{
      		savePressed=0; 
      		showSaved=0;
     	}
    }
    
	if(savePressed==0)
    {
		viewer.updateText("Press SHIFT+S to save",20,0,15,10,10,255,"saved");  
    }
 
}



void keyboardEventOccurred (const pcl::visualization::KeyboardEvent &event,void* taraxl3d_void)
{

	TaraXLPointcloud *taraxl3d = static_cast<TaraXLPointcloud *> (taraxl3d_void);

  	TARAXL_STATUS_CODE status;

  	if ((event.getKeySym () == "M" || event.getKeySym () =="m") && event.keyUp())
  	{
		switch (pcl_quality)
	    {
    		case quality::HIGHEST:
     		status = taraxl3d->setPointcloudQuality(TaraXLSDK::STANDARD);
     		if (status != TARAXL_SUCCESS)
     		{
      			cout << "Quality set failed" << endl;
      			break ;
     		}
     		pcl_quality = quality::STANDARD;
     		break;

     		case quality::MEDIUM:
     		status = taraxl3d->setPointcloudQuality(TaraXLSDK::HIGHEST);
     		if (status != TARAXL_SUCCESS)
     		{
      			cout << "Quality set failed" << endl;
     			 break ;
     		}
    		pcl_quality = quality::HIGHEST;
    		break;

     		case quality::STANDARD:
     		status = taraxl3d->setPointcloudQuality(TaraXLSDK::MEDIUM);
     		if (status != TARAXL_SUCCESS)
    		{
      			cout << "Quality set failed" << endl;
      			break ;
     		}
     		pcl_quality = quality::MEDIUM;
     		break;
	}

}


	if ((event.getKeySym () == "N" || event.getKeySym () =="n") && event.keyUp())
 	{
		switch (pcl_save)
   	 	{
			case PLY :
     		pcl_save = PCD;
     		break;

     		case PCD:
     		pcl_save = VTK;
     		break;

     		case VTK:
     		pcl_save = PLY;
     		break;
    	}
 	}


	if ( event.isShiftPressed() && (event.getKeySym () == "S" || event.getKeySym () =="s") && event.keyUp())
 	{
		ioMutex.lock();

		switch(pcl_save)
  		{
			case PLY :
     			status = taraxl3d->savePoints(TARAXL_PLY_CLOUD,"samplePLYCloud.ply"); 
     
     			if (status != TARAXL_SUCCESS)
    			{
      				cout << "Save failed" << endl;
      				break;
     			} 
     			savePressed=1;
     			break;
  

     		case PCD:
				status = taraxl3d->savePoints(TARAXL_PCD_CLOUD,"samplePCDCloud.pcd");
				if (status != TARAXL_SUCCESS)
				{
					cout << "Save failed" << endl;
			 		break;
				}
				savePressed=1;
				break;

			case VTK:
				status = taraxl3d->savePoints(TARAXL_VTK_CLOUD,"sampleVTKCloud.vtk");
				if (status != TARAXL_SUCCESS)
				{
					cout << "Save failed" << endl;
			 		break;
				}
				savePressed=1;
				break;
		 }

		 ioMutex.unlock();

	}

}


int main () 
{

	TaraXL taraxlCam;
  	TaraXLCam selectedCam;
 	TaraXLCamList taraxlCamList;
	ResolutionList supportedResolutions;
	TaraXLPointcloud *taraxl3d;
	pcl::visualization::CloudViewer *pclViewer;


	uint camIndex, iResIndex, iAccuracyMode;
	TARAXL_STATUS_CODE status;

	status = taraxlCam.enumerateDevices(taraxlCamList);
  	if (status != TARAXL_SUCCESS) 
	{
		cout << "Camera enumeration failed" << endl;
      	return 1;
  	}

  	if (taraxlCamList.size() == 0) 
	{
		cout << "No cameras connected" << endl;
    	return 1;
  	}

  	cout << "Select a device:" << endl;
  	for (int i = 0; i < taraxlCamList.size(); i++) 
	{
		cout << i << ": ";
    	string name;
    	taraxlCamList[i].getFriendlyName(name);
    	cout << name << endl;
  	}

  	cin >> camIndex;
  	if (cin.fail()) 
	{
		cout << "Invalid input" << endl;
    	return 1;
  	}

  	if (camIndex >= taraxlCamList.size()) 
	{
		cout << "Invalid input" << endl;
    	return 1;
  	}

  	selectedCam = taraxlCamList.at(camIndex);

  	status = selectedCam.connect();
  	if (status != TARAXL_SUCCESS) 
	{
		cout << "Camera connect failed" << endl;
      	return 1;
  	}

  	status = selectedCam.getResolutionList(supportedResolutions);
 	if (status != TARAXL_SUCCESS) 
	{
		cout << "Get camera resolutions failed" << endl;
      	return 1;
  	}

  	cout << endl << "Select a resolution:" << endl;
  	for (int i = 0; i < supportedResolutions.size(); i++) 
	{
		cout << i << ": ";
    	string resolution = "";
    	resolution += to_string(supportedResolutions[i].width);
    	resolution += " x ";
    	resolution += to_string(supportedResolutions[i].height);
    	cout << resolution << endl;
  	}
  	cin >> iResIndex;

  	if (cin.fail()) 
	{
		cout << "Invalid input" << endl;
	    return 1;
  	}

  	if (iResIndex >= supportedResolutions.size()) 
	{
    	cout << "Invalid input" << endl;
    	return 1;
  	}

  	status =selectedCam.setResolution(supportedResolutions.at(iResIndex));
  	if (status != TARAXL_SUCCESS) 
	{
		cout << "Set resolutions failed" << endl;
      	return 1;
  	}

  	taraxl3d = new TaraXLPointcloud(selectedCam);
  	if (taraxl3d == NULL) 
	{
		cout << "Unable to create instance to TaraDepth" << endl;
    	return 1;
  	}

  	cout<< endl << " \nPress m/M to cycle through the quality modes(HIGHEST,MEDIUM,STANDARD)"<<endl;
  	cout<< endl << " \nPress n/N to cycle through the PointCloud save formats(PLY,PCD,VTK)  "<<endl;
  	cout<< endl << " \nPress SHIFT+S to save the PointCloud in the current save format "<<endl;

        Points::Ptr currentCloud (new Points);



	cout << endl << "Loading pointcloud... " << endl;
	//Skipping initial 15 frames to allow the auto exposure to settle.
	for(int i = 0 ; i < 15 ; i++)
        	status =  taraxl3d->getPoints(currentCloud);


	bool init = true;

	Eigen::Affine3f Transform_Matrix = Eigen::Affine3f::Identity();

        float Trans_x = 0.0;
        float Trans_y = 0.0;
        float Trans_z = 0.0 ; //15.0;
        float Rot_x = 0.0;
        float Rot_y = 0.0;
        float Rot_z = 0.0;

        // Define a translation of 2.5 meters on the x axis.
        Transform_Matrix.translation() << Trans_x, Trans_y, Trans_z;

        // The same rotation matrix as before; tetha radians arround Z axis
        Transform_Matrix.rotate (Eigen::AngleAxisf (Rot_x, Eigen::Vector3f::UnitX()));
        Transform_Matrix.rotate (Eigen::AngleAxisf (Rot_y, Eigen::Vector3f::UnitY()));
        Transform_Matrix.rotate (Eigen::AngleAxisf (Rot_z, Eigen::Vector3f::UnitZ()));

        PointCloud<PointXYZRGB>::Ptr point_cloud_Transformed_ptr (new PointCloud<PointXYZRGB>);

        transformPointCloud (*currentCloud, *point_cloud_Transformed_ptr, Transform_Matrix);

	//Cloud viewer created
	pclViewer = new pcl::visualization::CloudViewer("TaraXL Point Cloud  Viewer");

        pclViewer->showCloud(currentCloud);
        cout << endl << "Pointcloud loaded successfully!" << endl;

        pclViewer->registerKeyboardCallback (keyboardEventOccurred, (void*)taraxl3d );

        pclViewer->runOnVisualizationThreadOnce (viewerOneOff);
        pclViewer->runOnVisualizationThread (viewerUpdate);
	
   	while(!pclViewer->wasStopped())
   	{
	
    	ioMutex.lock();
    	status =  taraxl3d->getPoints(currentCloud);
	
		if (status != TARAXL_SUCCESS) 
		{
			cout << "Get points failed" << endl;
      		return 1;
  		}
	
    	ioMutex.unlock();

	transformPointCloud (*currentCloud, *point_cloud_Transformed_ptr, Transform_Matrix);
        pclViewer->showCloud(point_cloud_Transformed_ptr);

	}

	delete taraxl3d;
	selectedCam.disconnect();
   	exit(0);
}
