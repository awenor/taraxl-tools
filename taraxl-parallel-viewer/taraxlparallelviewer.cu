#include <iostream>
#include <ctime>
#include  <iomanip>
#include <sys/time.h>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>

#include "TaraXL.h"
#include "TaraXLCam.h"
#include "TaraXLDepth.h"
#include<thread>
using namespace std;
using namespace cv;
using namespace TaraXLSDK;

std::thread lut1,lut2,lut3;

void lutR(Mat disp0,Mat lookUpTable_R,Mat &cdr)
{
  LUT(disp0,lookUpTable_R , cdr);
}

void lutG(Mat disp0,Mat lookUpTable_G,Mat &cdg)
{
  LUT(disp0,lookUpTable_G , cdg);
}

void lutB(Mat disp0,Mat lookUpTable_B,Mat &cdb)
{
  LUT(disp0,lookUpTable_B , cdb);
}

int main () {

  TaraXL taraxlCam;
  TaraXLCamList taraxlCamList;
  ResolutionList supportedResolutions;
  ACCURACY_MODE selectedMode;

  uint camIndex, iResIndex, iAccuracyMode;
  TARAXL_STATUS_CODE status;

  status = taraxlCam.enumerateDevices(taraxlCamList);
  if (status != TARAXL_SUCCESS) {

      cout << "Camera enumeration failed" << endl;
      return 1;
  }

  if (taraxlCamList.size() == 0) {

    cout << "No cameras connected" << endl;
    return 1;
  }
  cout << endl << "Select a Accuracy mode:" << endl;
   cout << "0: High Accuracy" << endl <<"1: Low Accuracy "<<endl<<"2: Ultra Accuracy" << endl;
   cin >> iAccuracyMode;

   if (cin.fail()) {

     cout << "Invalid input" << endl;
     return 1;
   }

   if (iAccuracyMode == 0) {

     selectedMode = HIGH;
   }
   else if (iAccuracyMode == 1) {

     selectedMode = LOW;
   }
   else if (iAccuracyMode == 2) {

     selectedMode = ULTRA;
   }

   else {

     cout << "Invalid input" << endl;
     return 1;
   }

  vector<Ptr<TaraXLDepth> > taraxlDepthList;
  vector<Mat> left, right, grayDisp, colorDisp, depthMap;
  TaraXLCam *selectedCam;
  vector<string> cameraUniqueIdList;
  for(int i = 0 ; i < taraxlCamList.size() ; i++)
  {
	  //selectedCam = new TaraXLCam(taraxlCamList.at(i));
	  status = taraxlCamList.at(i).connect();
	  if (status != TARAXL_SUCCESS) {

	      cout << "Camera connect failed " << status << endl;
	      return 1;
	  }
	  Ptr<TaraXLDepth> depth;
	  cout << "Camera connect status" << status << endl;
	  depth = new TaraXLDepth(taraxlCamList.at(i));
  	  if (depth == NULL)
	  {
    		cout << "Unable to create instance to TaraDepth" << endl;
    		return 1;
  	  }
	  depth->setAccuracy(selectedMode);
	  taraxlDepthList.push_back(depth);
          string id;
          taraxlCamList.at(i).getCameraUniqueId(id);
	  cameraUniqueIdList.push_back(id);
	  string windowName = "CAMERA : "+ id;
	  namedWindow(windowName, CV_WINDOW_AUTOSIZE);

	  Mat sample;
	  left.push_back(sample);
	  right.push_back(sample);
	  grayDisp.push_back(sample);
	  colorDisp.push_back(sample);
	  depthMap.push_back(sample);
  }

  Mat cdr, cdg, cdb;
  //COLORMAP JET LUT

  uchar r[] = {128,136,144,152,160,168,176,184,192,200,208,216,224,232,240,248,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,254,246,238,230,222,214,206,198,190,182,174,166,158,150,142,134,126,118,110,102,94,86,78,70,62,54,46,38,30,22,14,6,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};

  uchar g[] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,8,16,24,32,40,48,56,64,72,80,88,96,104,112,120,128,136,144,152,160,168,176,184,192,200,208,216,224,232,240,248,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,252,244,236,228,220,212,204,196,188,180,172,164,156,148,140,132,124,116,108,100,92,84,76,68,60,52,44,36,28,20,12,4,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};

  uchar b[] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,2,10,18,26,34,42,50,58,66,74,82,90,98,106,114,122,130,138,146,154,162,170,178,186,194,202,210,218,226,234,242,250,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,255,252,244,236,228,220,212,204,196,188,180,172,164,156,148,140,132};

  int minDisp,maxDisp;
  taraxlDepthList.at(0)->getMinDisparity(minDisp);

  uchar r1[256],g1[256],b1[256];

  for(int i = 0; i < 256 ; i++)
  {
        if(iAccuracyMode == 0)
        {
                if(i < 64)
                {
                        r1[i] = r1[i+1] = r[i];
                        g1[i] = g1[i+1] = g[i];
                        b1[i] = b1[i+1] = b[i];
                        i++;
                }
                else
                {
                        r1[i] = r[i-32];
                        g1[i] = g[i-32];
                        b1[i] = b[i-32];
                }
        }
        else
        {
                if(i <= minDisp)
                {
                        r1[i] = 0;
                        g1[i] = 0;
                        b1[i] = 0;
                }
                else
                {
                        r1[i] = r[i-minDisp];
                        g1[i] = g[i-minDisp];
                        b1[i] = b[i-minDisp];
                }
        }
  }

  Mat lookUpTable_R(1, 256, CV_8U,&r1), lookUpTable_G(1, 256, CV_8U,&g1), lookUpTable_B(1, 256, CV_8U,&b1);
  while(1)
  {
	for(int i = 0 ; i < taraxlCamList.size() ; i++)
	{

		status = taraxlDepthList.at(i)->getMap(left.at(i), right.at(i), grayDisp.at(i), true, depthMap.at(i), false, TARAXL_DEFAULT_FILTER);
		if (status != TARAXL_SUCCESS)
		{
			cout << "Get map failed" << endl;
			delete taraxlDepthList.at(i);
		        return 1;
		}
		grayDisp.at(i).convertTo(grayDisp.at(i),CV_8U);


    if(lut3.joinable())
    lut3.join();
    if(lut2.joinable())
    lut2.join();
    if(lut1.joinable())
    lut1.join();

    lut3 = std::thread(lutB, grayDisp.at(i),lookUpTable_B, std::ref(cdb));
    lut1 = std::thread(lutR, grayDisp.at(i),lookUpTable_R,std::ref(cdr));
    lut2 = std::thread(lutG, grayDisp.at(i),lookUpTable_G,std::ref( cdg));
    std::vector<cv::Mat> planes;
    if(lut3.joinable())
    lut3.join();
    planes.push_back(cdb);
    if(lut2.joinable())
    lut2.join();
    planes.push_back(cdg);
    if(lut1.joinable())
    lut1.join();
    planes.push_back(cdr);
    cv::merge(planes,colorDisp.at(i));

    cv::cvtColor(colorDisp.at(i), colorDisp.at(i), CV_RGBA2BGRA);


		string windowName = "CAMERA : "+ cameraUniqueIdList.at(i);
		imshow(windowName, colorDisp.at(i));
		waitKey(1);
	}
  }
  exit(0);
}
